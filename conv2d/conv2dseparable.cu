#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <helper_timer.h>
#include <hip/hip_runtime_api.h>

#define BLOCKDIM 256
#define STEP 4
#define MAXKernelRadius 32

__constant__ float c_kernel[64 + 1];

__global__ void conv2d_row(float *d_input, float *d_output, int img_w, int img_h, int kernelradius){
    
    extern __shared__ float s_data[];

    int idx_x = blockIdx.x * blockDim.x * STEP + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    int src = idx_y * img_w + idx_x;
    int dst = threadIdx.x + kernelradius + (blockDim.x + 2 * kernelradius ) * threadIdx.y;

    #pragma unroll
    for(int i = 0 ; i < STEP; i++){
        s_data[dst + i * blockDim.x] = d_input[src + i * blockDim.x];
    }

    if (threadIdx.x < kernelradius){
        if (idx_x < kernelradius){
            s_data[threadIdx.x + threadIdx.y * (2 * kernelradius + blockDim.x)] = 0;
        } else {
            s_data[threadIdx.x + threadIdx.y * (2 * kernelradius + blockDim.x)] = 0;
        }
    }


    if (blockDim.x - threadIdx.x < kernelradius)
    if (img_w - idx_x < kernelradius){
        s_data[(2 * kernelradius + blockDim.x) - threadIdx.x + threadIdx.y * (2 * kernelradius + blockDim.x)] = 0;
    }

    __syncthreads;
};
__global__ void conv2d_col(float *d_input, float *d_output, int img_w, int img_h, int kernelradius){


};

// add err checking
void processing(float* h_input, float *h_output, float *h_kernel, int img_w, int img_h, int kernelradius){
    // variables for device
    float *d_input, *d_intermediate_output, *d_output, *d_kernel;
    int buf_size = img_w * img_h * sizeof(float);
    int kernel_size = (kernelradius * 2 + 1) * sizeof(float);

    StopWatchInterface *timer;
    sdkCreateTimer(&timer);

    // allocate device mem
    hipMalloc((void **)&d_input, buf_size);
    hipMalloc((void **)&d_intermediate_output, buf_size);
    hipMalloc((void **)&d_output, buf_size);

    // send data
    hipMemcpy(d_input, h_input, buf_size, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_kernel), h_kernel, kernel_size);

    // dimensions
    int temp = STEP * BLOCKDIM;
    dim3 dimBlock(BLOCKDIM, BLOCKDIM);
    dim3 dimGrid_row((img_w + temp - 1)/temp, (img_h + BLOCKDIM - 1)/BLOCKDIM);
    int shared_mem_size = BLOCKDIM * (BLOCKDIM * STEP + 2 * kernelradius) * sizeof(float);

    // where magic happens
    // row
    sdkStartTimer(&timer);
    hipProfilerStart();
    conv2d_row<<<dimGrid_row, dimBlock, shared_mem_size, 0>>>(d_input, d_intermediate_output, img_w, img_h, kernelradius);
    hipProfilerStop();
    sdkStopTimer(&timer);
    printf("Processing Time: %.2f ms\n", sdkGetTimerValue(&timer));

    dim3 dimGrid_col((img_w + BLOCKDIM - 1)/BLOCKDIM, (img_h + temp - 1)/temp);
    
    //col
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);
    hipProfilerStart();
    conv2d_col<<<dimGrid_col, dimBlock, shared_mem_size, 0>>>(d_intermediate_output, d_output, img_w, img_h, kernelradius);
    hipProfilerStop();
    sdkStopTimer(&timer);
    printf("Processing Time: %.2f ms\n", sdkGetTimerValue(&timer));
    
    hipDeviceSynchronize();

    // return data
    hipMemcpy(h_output, d_output, buf_size, hipMemcpyDeviceToHost);

    // free
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_intermediate_output);

}
