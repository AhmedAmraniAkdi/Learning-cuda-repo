
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

#define IMROWS 1204
#define IMCOLS 1880 
#define IMCHANNELS 1
#define KERNELRADIUS 16 // 16x2 + 1
#define BLOCKSIZE 16

__constant__ float c_kernel[KERNELRADIUS * 2 + 1];

void processing(float* h_input, float *h_output, float *h_kernel){
    // variables for device
    float* *d_input,*d_output, *d_kernel;
    int buf_size = IMROWS * IMCOLS * sizeof(float);
    int kernel_size = (KERNELRADIUS*2+1) * sizeof(float);

    // allocate device mem
    hipMalloc((void **)&d_input, buf_size);
    hipMalloc((void **)&d_output, buf_size);
    hipMalloc((void **)&d_kernel, kernel_size);

    // send data
    hipMemcpy(d_input, h_input, buf_size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, h_kernel, kernel_size, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_kernel), h_kernel, kernel_size);



    
}
