#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>


#define ARR_SIZE (1 << 20)
#define BLOCKSIZE1 (1 << 5)
#define GRIDSIZE1 (1 << 15)
#define LOG2BLOCKSIZE1 5
#define LOG2ARR_SIZE 20

#define BLOCKSIZE2 (1 << 5)
#define GRIDSIZE2 (1 << 8)
#define STRIDE_THREAD 16 // each thread processes 16 elements
#define STRIDE_BLOCK (1 << 11) // each block processes N/gridsize elements/2 elements of A and elements of B (arrs to merge)
#define SMEM_SIZE 256
#define LOG2SMEM 8
#define REFILL_LOADS 8 // times we have to refill SMEM = 2^11 elements / 256
#define SMEM_LOADS 8 // 32 threads, 256 smem, need to read 8 times

/*
    merge path configuration:

    ok, hear me out on this one :

    256 blocks of 32 threads
    
    64Kb of smem per sm, 5 sm, 
    we will take 512 smem elems per block making it 256 elements of A, and 256 elements of B

    we will take a stride of 16 elements per thread

    0) first iteration
        we start merging arrays of length 32 , that makes it 2^14 pairs to sort
        2^6 pairs per block
        2^11 (x32) elements of A's and 2^11 elements of B's - at 256 elements smem's 
        -> 8 loads of A's and B's and 8 pairs fit per iter (256/32)

        512 total elements on 1 load / 32 threads = 16 elements per thread = stride -> all smem consumed on 1 iteration

        64 total elements(A+B) / 16 elements per thread = 4 threads per pair
        -> 8 pairs total (32 threads)

    1) second iteration
        length 64, 2^13 pairs
        2^5 pairs per block
        2^11 elements of A's and 2^11 B's, 8 loads

        128 total elements(A+B) / 16 elements per thread = 8 threads per pair
        -> 4 pairs total

    ...

    at 8) we start will need 2 blocks for a pair -> will need grid_partition_path

*/


/*

    Problem: we don't know which 512 elements the block is processing, could be 512 from A and 0 from B, 256/256, 100/412 ...
    we can't just load 256/256 each time... 

    solution: have 1 array of smem with 512 size, we fill it, keep track where A ends and B starts and keep track for the next offsets x, y for the next iteration
    we could have both A, B 512 size smem arrays but we waste smem + could possibly go over the smem limit

    this doesn't change the calculations done in the previous comment.

    But!!!! yes, we have where to start loading but ... when to stop?

    what we will do is: 
    
    1-for L sized segment we need at most L of A and at most B L, but only L elements will be consumed, yes?
    we load L from A, L from B, the last thread gives us the new starting point and we might read the same data again.

    2-we can also use grid_partition_path but we increase A_diag and B_diag and find diagonal point for each smem size

    which one is better?

    1- L sized path needs L elements, we are reading 2L elements, so we will be rereading L elements every time but on coalesced manner!
    + only starts becoming an issue when the arrays to merge don't fit completly on smem, so size 512 A and 512 B
    + possibility to just move the unused items to the start of the arrays instead of rereading (*)
    + check for when either A or B are fully consumed and just start filling with the unfinished array?
    let's say we don't do (*) for 512 sized array, we have 2^10 total pairs, for each pair we read 2*512 elements again, 
    which means we reread the whole array again so N reads - it's 4 am, maybe there is a mistake, but the logic is correct... i believe

    2- A_diag and B_diag become N/SMEM = 2^20/2^9 from 2^8, that's 8 times more and we have to call this kernel many times more
    reads are reaaaly uncoalesced, which means they are serialised -> x32 the number of each warpsize read
    + only starts becoming an issue when the arrays to merge don't fit completly on smem, so size 512 A and 512 B
    paper said we need at most log (size Array) to find partition point ... so total 2^11 * 20 * 32 .. bigger than (1)

    we will stick to 1 and move elements instead of refilling.

*/

// (**) reminder, check for when either A or B are fully consumed and then just directly merge the remaining items of the other array

__device__ void seq_merge(float *dest, float *A, int start_a, int end_a, float *B, int start_b, int end_b){

    float item_A = A[start_a];
    float item_B = B[start_b];

    #pragma unroll
    for(int i = 0; i < STRIDE_THREAD; i++){

        bool p = (start_b < end_b) && ((start_a >= end_a) || item_B >= item_A);

        if(p){
            dest[i] = item_B; 
            item_B = B[++start_b];  
        } else {
            dest[i] = item_A;
            item_A = B[++start_a]; 
        }

    }
    
}

/*

    we will have 3 similar merge sorts :
    1- for when arrays A, B fit into smem so size <256
    2- arrays A and B don't fit so size >=512
    3- when size is > 2^11, 2 or more blocks needed for the A and B

    why? keep code cleaner

*/


__global__ void merge_sort_small(float *d_input, int length){

    __shared__ float A[SMEM_SIZE + 1]; // stride 16, so thread 0 reades bank 0, thread 1 at bank 16, thread 2 at bank 0,
    __shared__ float B[SMEM_SIZE + 1]; // bank conflict, +1 padding for no conflicts, t0 0, t1 16, t2 1 ... accessing same index doesn't cause conflict
    __shared__ float Out[SMEM_SIZE + 1]; // for coalesced writing

    int idx = blockIdx.x * STRIDE_BLOCK + threadIdx.x;

}

__global__ void merge_sort_medium(float *d_input, int length){

    __shared__ float A[SMEM_SIZE + 1]; // stride 16, so thread 0 reades bank 0, thread 1 at bank 16, thread 2 at bank 0,
    __shared__ float B[SMEM_SIZE + 1]; // bank conflict, +1 padding for no conflicts, t0 0, t1 16, t2 1 ... accessing same index doesn't cause conflict
    __shared__ float Out[SMEM_SIZE + 1]; // for coalesced writing

    int idx = blockIdx.x * STRIDE_BLOCK + threadIdx.x;

}

__global__ void merge_sort_large(float *d_input, int length, int *diag_A, int *diag_B){

    __shared__ float A[SMEM_SIZE + 1]; // stride 16, so thread 0 reades bank 0, thread 1 at bank 16, thread 2 at bank 0,
    __shared__ float B[SMEM_SIZE + 1]; // bank conflict, +1 padding for no conflicts, t0 0, t1 16, t2 1 ... accessing same index doesn't cause conflict
    __shared__ float Out[SMEM_SIZE + 1]; // for coalesced writing

    int idx = blockIdx.x * STRIDE_BLOCK + threadIdx.x;

}

/*
    ok, so what's the problem... imagine we are merging 2 sorted arrays A and B...
    and what a single block processes is less than the length of each array...
    we will need for example 2 blocks to merge the 2 arrays...
    block 1 will start from the top left corner finding each intersection of the diagonals with the path
    but block 2 does start where? what's the x and y offsets? we can't have communication between blocks

    the solution: make a gridsize partition when merging, that way each block has it own x, y offset

    inconvenient: we can't use shared memory: the number of elements is too large for it
    convenient: A diag and B diag are gridsize arrays, so small
*/

// gets called onyl when more than 1 block is needed to process the arrays
// 1 block 256 threads
__global__ void grid_partition_path(float *d_input, int length, int *diag_A, int *diag_B, int blocksperarray){

    // get where in d_input we are
    d_input += threadIdx.x * STRIDE_BLOCK;
    
    float *A = d_input;
    float *B = d_input + length;
    
    // blocksperarray blocks process the array
    // so each blocksperarray_i block starts at 0
    if(threadIdx.x & (blocksperarray - 1)){
        diag_A[threadIdx.x] = 0;
        diag_B[threadIdx.x] = 0;
    } else {
    
        int diag = (threadIdx.x + 1) * length * 2 / blockDim.x;
        int atop = diag > length ? length : diag;
        int btop = diag > length ? diag - length : 0;
        int abot = btop;

        int ai, bi;
        int offset;

        while(1){

            offset = (atop - abot)/2;
            ai = atop - offset;
            bi = btop + offset;

            if (ai >= 0 && bi <= length && (A[ai] > B[bi - 1] || ai == length || bi == 0)){
                if((A[ai - 1] <= B[bi] || ai == 0 || bi == length)){
                    diag_A[threadIdx.x] = ai;
                    diag_B[threadIdx.x] = bi;
                } else {
                    atop = ai - 1;
                    btop = bi + 1; 
                }
            } else {
                abot = ai + 1;
            }
        }
    }

}

/*
__global__ void odd_even_merge_sort(float *d_input){

    __shared__ float s_data[BLOCKSIZE1];

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    s_data[threadIdx.x] = d_input[idx];

    __syncthreads();

    int temp;

    #pragma unroll
    for(int p = 1 << (LOG2BLOCKSIZE1 - 1); p > 0; p /= 2) {
        
        int q = 1 << LOG2BLOCKSIZE1;
        int r = 0;

        #pragma unroll
        for (int d = p ; d > 0 ; d = q - p) {

            if(threadIdx.x < BLOCKSIZE1 - d){
            
                if ((threadIdx.x & p) == r) {
                    if (s_data[threadIdx.x] > s_data[threadIdx.x + d]){
                        temp = s_data[threadIdx.x];
                        s_data[threadIdx.x] = s_data[threadIdx.x + d];
                        s_data[threadIdx.x + d] = temp;
                    }
                }
            }

            q /= 2;
            r = p;

            __syncthreads();
        }

    }

    d_input[idx] = s_data[threadIdx.x];

}
*/



/*
    starting with the merge path from length 1 arrays is a bit overkill...
    what we do is a bitonic sort getting a collection of 32 size sorted arrays, we start merging these
    we will need log N - log 32 merging steps
    why 32? fits nicely with the warpsize - no synchronisation needed and gives us ability to use warp shuffle functions

*/

__device__ float swap(int x, int mask, int dir){
    float y = __shfl_xor_sync(0xffffffff, x, mask);
    return x < y == dir ? y : x;
}

__device__ unsigned int bfe(unsigned int x, unsigned int bit, unsigned int num_bits=1){
    return (x >> bit) & 1;
}

// x0 > x1
// thread 0;  x0; x0 = swap(x0, 1, 0)  ; y = get(xi from 0^1=1) = x1 ; return x0 < x1 == 0 ? x1 : x0 -> x1
// thread 1;  x1; x1 = swap(x1, 1, 1)  ; y = get(x1 from 1^1=0) = x0 ; return x1 < x0 == 1 ? x0 : x1 -> x0

__global__ void warpsize_bitonic_sort(float *d_input){

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    float x = d_input[idx];

    x = swap(x, 0x01, bfe(threadIdx.x, 1) ^ bfe(threadIdx.x, 0)); // 2
    x = swap(x, 0x02, bfe(threadIdx.x, 2) ^ bfe(threadIdx.x, 1)); // 4
    x = swap(x, 0x01, bfe(threadIdx.x, 2) ^ bfe(threadIdx.x, 0));
    x = swap(x, 0x04, bfe(threadIdx.x, 3) ^ bfe(threadIdx.x, 2)); // 8
    x = swap(x, 0x02, bfe(threadIdx.x, 3) ^ bfe(threadIdx.x, 1));
    x = swap(x, 0x01, bfe(threadIdx.x, 3) ^ bfe(threadIdx.x, 0));
    x = swap(x, 0x08, bfe(threadIdx.x, 4) ^ bfe(threadIdx.x, 3)); // 16
    x = swap(x, 0x04, bfe(threadIdx.x, 4) ^ bfe(threadIdx.x, 2));
    x = swap(x, 0x02, bfe(threadIdx.x, 4) ^ bfe(threadIdx.x, 1));
    x = swap(x, 0x01, bfe(threadIdx.x, 4) ^ bfe(threadIdx.x, 0));
    x = swap(x, 0x10, bfe(threadIdx.x, 5) ^ bfe(threadIdx.x, 4)); // 32
    x = swap(x, 0x08, bfe(threadIdx.x, 5) ^ bfe(threadIdx.x, 3));
    x = swap(x, 0x04, bfe(threadIdx.x, 5) ^ bfe(threadIdx.x, 2));
    x = swap(x, 0x02, bfe(threadIdx.x, 5) ^ bfe(threadIdx.x, 1));
    x = swap(x, 0x01, bfe(threadIdx.x, 5) ^ bfe(threadIdx.x, 0));

    d_input[idx] = x;
}

// main + interface
void cuda_interface_sort(float* d_input){

    dim3 dimBlock(BLOCKSIZE1);
    dim3 dimGrid(GRIDSIZE1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsed_time;
    float total_time = 0;

    hipEventRecord(start, 0);
    //odd_even_merge_sort<<<dimGrid, dimBlock>>>(d_input);
    warpsize_bitonic_sort<<<dimGrid, dimBlock>>>(d_input);
    checkCudaErrors(hipGetLastError());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    total_time += elapsed_time;
    printf( "warpsize bitonic sort: %.8f ms\n", elapsed_time);


    int *diag_A, *diag_B;
    hipMalloc((void **)&diag_A, GRIDSIZE2 * sizeof(float));
    hipMalloc((void **)&diag_B, GRIDSIZE2 * sizeof(float));

    hipEventRecord(start, 0);
    
    // 32 -> 256 (inclusive)
    for(int i = LOG2BLOCKSIZE1; i <= LOG2SMEM; i++){
        merge_sort_small<<<GRIDSIZE2, BLOCKSIZE2>>>(d_input, 1 << i);
    }
    // 256 -> 1024 (inclusive)
    for(int i = LOG2SMEM + 1; i <= 10; i++){
        merge_sort_medium<<<GRIDSIZE2, BLOCKSIZE2>>>(d_input, 1 << i);
    }
    // 2048 -> N/2
    int blocksperarray = 2;
    for(int i = STRIDE_BLOCK; i <= LOG2ARR_SIZE - 1; i++){
        grid_partition_path<<<1, GRIDSIZE2>>>(d_input, 1 << i, diag_A, diag_B, blocksperarray);
        merge_sort_large<<<GRIDSIZE2, BLOCKSIZE2>>>(d_input, 1 << i, diag_A, diag_B);
        blocksperarray <<= 1;
    }

    //merge_sort<<<GRIDSIZE2, BLOCKSIZE2>>>(d_input, 0, diag_A, diag_B);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    total_time += elapsed_time;
    printf( "merge sort using merge path: %.8f ms\n", elapsed_time);

    printf("total time:%f\n", total_time);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}   

void fill_array(float *h_input){

    for(int i = 0; i <  ARR_SIZE; i++){
        h_input[i] = (float) (rand() & 255);
    }
}

int main(void){

    srand(0);

    float *h_input;
    float *d_input;

    h_input = (float*) malloc(ARR_SIZE * sizeof(float));
     
    fill_array(h_input);

    hipMalloc((void **)&d_input, ARR_SIZE * sizeof(float));
    checkCudaErrors(hipGetLastError());
    
    hipMemcpy(d_input, h_input, ARR_SIZE * sizeof(float), hipMemcpyHostToDevice);

    cuda_interface_sort(d_input);

    hipMemcpy(h_input, d_input,  ARR_SIZE * sizeof(float), hipMemcpyDeviceToHost);
      
    checkCudaErrors(hipGetLastError());

    for(int i = 0; i < 64; i++){
        printf("%f ", h_input[i]);
        if(i == 31){
            printf("\n");
        }
    }
    hipFree(d_input);
    free(h_input);

    return 0;
}