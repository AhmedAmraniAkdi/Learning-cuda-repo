#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>


#define ARR_SIZE (1 << 20)
#define BLOCKSIZE1 (1 << 5)
#define GRIDSIZE1 (1 << 15)
#define LOG2BLOCKSIZE1 5
#define LOG2ARR_SIZE 20

#define BLOCKSIZE2 (1 << 5)
#define GRIDSIZE2 (1 << 8)

/*
    merge path configuration:





*/


__device__ void seq_merge(float *dest, float *A, int start_a, int end_a, float *B, int start_b, int end_b){


    
}

__global__ void merge_sort(float *d_input, int length, float *diag_A, float *diag_B){



}


/*
    ok, so what's the problem... imagine we are merging 2 sorted arrays A and B...
    and what a single block processes is less than the length of each array...
    we will need for example 2 blocks to merge the 2 arrays...
    block 1 will start from the top left corner finding each intersection of the diagonals with the path
    but block 2 does start where? what's the x and y offsets? we can't have communication between blocks

    the solution: make a gridsize partition when merging, that way each block has it own x, y offset

    inconvenient: we can't use shared memory: the number of elements is too large for it
    convenient: A diag and B diag are gridsize arrays, so small
*/

// gets called onyl when more than 1 block is needed to process the arrays
// 1 block 256 threads
__global__ void grid_partition_path(float *d_input, float *diag_A, float *diag_B, int length, int blocksperarray){

    // get where in d_input we are
    //d_input += something;
    
    float *A = d_input;
    float *B = d_input + length;
    
    // blocksperarray blocks process the array
    // so each blocksperarray_i block starts at 0
    if(threadIdx.x & (blocksperarray - 1)){
        diag_A[threadIdx.x] = 0;
        diag_B[threadIdx.x] = 0;
    }
    
    int diag = (threadIdx.x + 1) * length * 2 / blockDim.x;
    int atop = diag > length ? length : diag;
    int btop = diag > length ? diag - length : 0;
    int abot = btop;

    int ai, bi;
    int offset;

    while(1){

        offset = (atop - abot)/2;
        ai = atop - offset;
        bi = btop + offset;

        if (A[ai] > B[bi - 1]){
            if(A[ai - 1] <= B[bi]){
                diag_A[threadIdx.x] = ai;
                diag_B[threadIdx.x] = bi;
            } else {
                atop = ai - 1;
                btop = bi + 1; 
            }
        } else {
            abot = ai + 1;
        }
    }

}

/*
__global__ void odd_even_merge_sort(float *d_input){

    __shared__ float s_data[BLOCKSIZE1];

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    s_data[threadIdx.x] = d_input[idx];

    __syncthreads();

    int temp;

    #pragma unroll
    for(int p = 1 << (LOG2BLOCKSIZE1 - 1); p > 0; p /= 2) {
        
        int q = 1 << LOG2BLOCKSIZE1;
        int r = 0;

        #pragma unroll
        for (int d = p ; d > 0 ; d = q - p) {

            if(threadIdx.x < BLOCKSIZE1 - d){
            
                if ((threadIdx.x & p) == r) {
                    if (s_data[threadIdx.x] > s_data[threadIdx.x + d]){
                        temp = s_data[threadIdx.x];
                        s_data[threadIdx.x] = s_data[threadIdx.x + d];
                        s_data[threadIdx.x + d] = temp;
                    }
                }
            }

            q /= 2;
            r = p;

            __syncthreads();
        }

    }

    d_input[idx] = s_data[threadIdx.x];

}
*/



/*
    starting with the merge path from length 1 arrays is a bit overkill...
    what we do is a bitonic sort getting a collection of 32 size sorted arrays, we start merging these
    we will need log N - log 32 merging steps
    why 32? fits nicely with the warpsize - no synchronisation needed and gives us ability to use warp shuffle functions

*/

__device__ float swap(int x, int mask, int dir){
    float y = __shfl_xor_sync(0xffffffff, x, mask);
    return x < y == dir ? y : x;
}

__device__ unsigned int bfe(unsigned int x, unsigned int bit, unsigned int num_bits=1){
    return (x >> bit) & 1;
}

// x0 > x1
// thread 0;  x0; x0 = swap(x0, 1, 0)  ; y = get(xi from 0^1=1) = x1 ; return x0 < x1 == 0 ? x1 : x0 -> x1
// thread 1;  x1; x1 = swap(x1, 1, 1)  ; y = get(x1 from 1^1=0) = x0 ; return x1 < x0 == 1 ? x0 : x1 -> x0

__global__ void warpsize_bitonic_sort(float *d_input){

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    float x = d_input[idx];

    x = swap(x, 0x01, bfe(threadIdx.x, 1) ^ bfe(threadIdx.x, 0)); // 2
    x = swap(x, 0x02, bfe(threadIdx.x, 2) ^ bfe(threadIdx.x, 1)); // 4
    x = swap(x, 0x01, bfe(threadIdx.x, 2) ^ bfe(threadIdx.x, 0));
    x = swap(x, 0x04, bfe(threadIdx.x, 3) ^ bfe(threadIdx.x, 2)); // 8
    x = swap(x, 0x02, bfe(threadIdx.x, 3) ^ bfe(threadIdx.x, 1));
    x = swap(x, 0x01, bfe(threadIdx.x, 3) ^ bfe(threadIdx.x, 0));
    x = swap(x, 0x08, bfe(threadIdx.x, 4) ^ bfe(threadIdx.x, 3)); // 16
    x = swap(x, 0x04, bfe(threadIdx.x, 4) ^ bfe(threadIdx.x, 2));
    x = swap(x, 0x02, bfe(threadIdx.x, 4) ^ bfe(threadIdx.x, 1));
    x = swap(x, 0x01, bfe(threadIdx.x, 4) ^ bfe(threadIdx.x, 0));
    x = swap(x, 0x10, bfe(threadIdx.x, 5) ^ bfe(threadIdx.x, 4)); // 32
    x = swap(x, 0x08, bfe(threadIdx.x, 5) ^ bfe(threadIdx.x, 3));
    x = swap(x, 0x04, bfe(threadIdx.x, 5) ^ bfe(threadIdx.x, 2));
    x = swap(x, 0x02, bfe(threadIdx.x, 5) ^ bfe(threadIdx.x, 1));
    x = swap(x, 0x01, bfe(threadIdx.x, 5) ^ bfe(threadIdx.x, 0));

    d_input[idx] = x;
}

// main + interface
void cuda_interface_sort(float* d_input){

    dim3 dimBlock(BLOCKSIZE1);
    dim3 dimGrid(GRIDSIZE1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsed_time;
    float total_time = 0;

    hipEventRecord(start, 0);
    //odd_even_merge_sort<<<dimGrid, dimBlock>>>(d_input);
    warpsize_bitonic_sort<<<dimGrid, dimBlock>>>(d_input);
    checkCudaErrors(hipGetLastError());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    total_time += elapsed_time;
    printf( "warpsize bitonic sort: %.8f ms\n", elapsed_time);


    float *diag_A, *diag_B;
    hipMalloc((void **)&diag_A, GRIDSIZE2 * sizeof(float));
    hipMalloc((void **)&diag_B, GRIDSIZE2 * sizeof(float));

    hipEventRecord(start, 0);

    for(int i = LOG2BLOCKSIZE1; i <= LOG2ARR_SIZE; i++){
        // do some if elses
        // grid partition
        // merge

    }
        //merge_sort<<<GRIDSIZE1, BLOCKSIZE1>>>(d_input, (1 << i));
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    total_time += elapsed_time;
    printf( "merge sort using merge path: %.8f ms\n", elapsed_time);

    printf("total time:%f\n", total_time);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}   

void fill_array(float *h_input){

    for(int i = 0; i <  ARR_SIZE; i++){
        h_input[i] = (float) (rand() & 255);
    }
}

int main(void){

    srand(0);

    float *h_input;
    float *d_input;

    h_input = (float*) malloc(ARR_SIZE * sizeof(float));
     
    fill_array(h_input);

    hipMalloc((void **)&d_input, ARR_SIZE * sizeof(float));
    checkCudaErrors(hipGetLastError());
    
    hipMemcpy(d_input, h_input, ARR_SIZE * sizeof(float), hipMemcpyHostToDevice);

    cuda_interface_sort(d_input);

    hipMemcpy(h_input, d_input,  ARR_SIZE * sizeof(float), hipMemcpyDeviceToHost);
      
    checkCudaErrors(hipGetLastError());

    for(int i = 0; i < 64; i++){
        printf("%f ", h_input[i]);
        if(i == 31){
            printf("\n");
        }
    }
    hipFree(d_input);
    free(h_input);

    return 0;
}