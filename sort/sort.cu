#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#define ARR_SIZE (1 << 25)

__global__ void merge_path(float *d_input){

}

__global__ void odd_even_merge_sort(float *d_input){

}

// main + interface
void cuda_interface_sort(float* d_input){

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsed_time;

    hipEventRecord(start, 0);


    // odd even + merge path on loop


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf( "scan: %.8f ms\n", elapsed_time);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}   

void fill_array(float *h_input){

    for(int i = 0; i <  ARR_SIZE; i++){
        h_input[i] = (float) rand();
    }
}

int main(void){

    srand(0);

    float *h_input;
    float *d_input;

    h_input = (float*) malloc(ARR_SIZE * sizeof(float));
     
    fill_array(h_input);

    hipMalloc((void **)&d_input, ARR_SIZE * sizeof(float));
    hipMemcpy(d_input, h_input, ARR_SIZE * sizeof(float), hipMemcpyHostToDevice);

    cuda_interface_sort(d_input);

    hipMemcpy(h_input, d_input,  ARR_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 0; i < 1024; i++)
        printf("sum %.8f", h_input[i]);

    hipFree(d_input);
    free(h_input);

    return 0;
}