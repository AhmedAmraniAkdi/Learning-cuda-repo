#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>


#define ARR_SIZE (1 << 20)
#define BLOCKSIZE1 (1 << 5)
#define GRIDSIZE1 (1 << 15)
#define LOG2BLOCKSIZE1 5
#define LOG2ARR_SIZE 20

#define BLOCKSIZE2 (1 << 64)
#define GRIDSIZE2 (1 << 10)

/*
__device__ void seq_merge(float *dest, float *A, int start_a, int end_a, float *B, int start_b, int end_b){


    
}

__global__ void merge_sort(float *d_input, int length, float *diag_A, float *diag_B){



}

__global__ void merge_path(float *d_input, int length, float *diag_A, float *diag_B){



}*/

/*
__global__ void odd_even_merge_sort(float *d_input){

    __shared__ float s_data[BLOCKSIZE1];

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    s_data[threadIdx.x] = d_input[idx];

    __syncthreads();

    int temp;

    #pragma unroll
    for(int p = 1 << (LOG2BLOCKSIZE1 - 1); p > 0; p /= 2) {
        
        int q = 1 << LOG2BLOCKSIZE1;
        int r = 0;

        #pragma unroll
        for (int d = p ; d > 0 ; d = q - p) {

            if(threadIdx.x < BLOCKSIZE1 - d){
            
                if ((threadIdx.x & p) == r) {
                    if (s_data[threadIdx.x] > s_data[threadIdx.x + d]){
                        temp = s_data[threadIdx.x];
                        s_data[threadIdx.x] = s_data[threadIdx.x + d];
                        s_data[threadIdx.x + d] = temp;
                    }
                }
            }

            q /= 2;
            r = p;

            __syncthreads();
        }

    }

    d_input[idx] = s_data[threadIdx.x];

}
*/

__device__ float swap(int x, int mask, int dir){
    float y = __shfl_xor_sync(0xffffffff, x, mask);
    return x < y == dir ? y : x;
}

__device__ unsigned int bfe(unsigned int x, unsigned int bit, unsigned int num_bits=1){
    return (x >> bit) & 1;
}

// x0 > x1
// thread 0;  x0; x0 = swap(x0, 1, 0)  ; y = get(xi from 0^1=1) = x1 ; return x0 < x1 == 0 ? x1 : x0 -> x1
// thread 1;  x1; x1 = swap(x1, 1, 1)  ; y = get(x1 from 1^1=0) = x0 ; return x1 < x0 == 1 ? x0 : x1 -> x0

__global__ void warpsize_bitonic_sort(float *d_input){

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    float x = d_input[idx];

    x = swap(x, 0x01, bfe(threadIdx.x, 1) ^ bfe(threadIdx.x, 0)); // 2
    x = swap(x, 0x02, bfe(threadIdx.x, 2) ^ bfe(threadIdx.x, 1)); // 4
    x = swap(x, 0x01, bfe(threadIdx.x, 2) ^ bfe(threadIdx.x, 0));
    x = swap(x, 0x04, bfe(threadIdx.x, 3) ^ bfe(threadIdx.x, 2)); // 8
    x = swap(x, 0x02, bfe(threadIdx.x, 3) ^ bfe(threadIdx.x, 1));
    x = swap(x, 0x01, bfe(threadIdx.x, 3) ^ bfe(threadIdx.x, 0));
    x = swap(x, 0x08, bfe(threadIdx.x, 4) ^ bfe(threadIdx.x, 3)); // 16
    x = swap(x, 0x04, bfe(threadIdx.x, 4) ^ bfe(threadIdx.x, 2));
    x = swap(x, 0x02, bfe(threadIdx.x, 4) ^ bfe(threadIdx.x, 1));
    x = swap(x, 0x01, bfe(threadIdx.x, 4) ^ bfe(threadIdx.x, 0));
    x = swap(x, 0x10, bfe(threadIdx.x, 5) ^ bfe(threadIdx.x, 4)); // 32
    x = swap(x, 0x08, bfe(threadIdx.x, 5) ^ bfe(threadIdx.x, 3));
    x = swap(x, 0x04, bfe(threadIdx.x, 5) ^ bfe(threadIdx.x, 2));
    x = swap(x, 0x02, bfe(threadIdx.x, 5) ^ bfe(threadIdx.x, 1));
    x = swap(x, 0x01, bfe(threadIdx.x, 5) ^ bfe(threadIdx.x, 0));

    d_input[idx] = x;
}

// main + interface
void cuda_interface_sort(float* d_input){

    dim3 dimBlock(BLOCKSIZE1);
    dim3 dimGrid(GRIDSIZE1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsed_time;
    float total_time = 0;

    hipEventRecord(start, 0);
    //odd_even_merge_sort<<<dimGrid, dimBlock>>>(d_input);
    warpsize_bitonic_sort<<<dimGrid, dimBlock>>>(d_input);
    checkCudaErrors(hipGetLastError());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    total_time += elapsed_time;
    printf( "warpsize bitonic sort: %.8f ms\n", elapsed_time);


    /*float *diag_A, *diag_B;
    hipMalloc((void **)&diag_A, BLOCKSIZE2 * GRIDSIZE2 * sizeof(float));
    hipMalloc((void **)&diag_B, BLOCKSIZE2 * GRIDSIZE2 * sizeof(float));*/

    /*hipEventRecord(start, 0);

    for(int i = LOG2BLOCKSIZE1; i <= LOG2ARR_SIZE; i++)
        merge_sort<<<GRIDSIZE1, BLOCKSIZE1>>>(d_input, (1 << i));
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    total_time += elapsed_time;
    printf( "merge sort using merge path: %.8f ms\n", elapsed_time);*/

    printf("total time:%f\n", total_time);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}   

void fill_array(float *h_input){

    for(int i = 0; i <  ARR_SIZE; i++){
        h_input[i] = (float) (rand() & 255);
    }
}

int main(void){

    srand(0);

    float *h_input;
    float *d_input;

    h_input = (float*) malloc(ARR_SIZE * sizeof(float));
     
    fill_array(h_input);

    hipMalloc((void **)&d_input, ARR_SIZE * sizeof(float));
    checkCudaErrors(hipGetLastError());
    
    hipMemcpy(d_input, h_input, ARR_SIZE * sizeof(float), hipMemcpyHostToDevice);

    cuda_interface_sort(d_input);

    hipMemcpy(h_input, d_input,  ARR_SIZE * sizeof(float), hipMemcpyDeviceToHost);
      
    checkCudaErrors(hipGetLastError());

    for(int i = 0; i < 64; i++){
        printf("%f ", h_input[i]);
        if(i == 31){
            printf("\n");
        }
    }
    hipFree(d_input);
    free(h_input);

    return 0;
}