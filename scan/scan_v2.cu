#include "hip/hip_runtime.h"
/*  Let's do better

    Efficient Parallel Scan Algorithms for GPUs
        Shubhabrata Sengupta Davis Mark Harris Michael Garland

    Parallel Scan for Stream Architectures1
        Duane Merrill Andrew Grimshaw

    Inclusive scan

    We need to beat the 12ms mark on 2^25 elements (~33m elements)

*/
 
// for now everything is power of 2, normally this won't be the case -> padding + if elses

#define ARR_SIZE (1 << 25)
#define BLOCKSIZE 512
#define LOG2_BLOCKSIZE 9
#define REDUCTION_STEPS 2 // each thread thread loads 2 float4, 128B
#define SCAN_STEPS REDUCTION_STEPS // each block thread loads 2 float4, 128B
#define SCAN_SMEM_WIDTH (BLOCKSIZE/32)
#define LOG2_SCAN_SMEM_WIDTH 4
#define MIDDLE_SCAN_STEP 16 // 2^(25 - 3 - 9 - 9) // -3 (2 float4 loads) - 9 (blocksize) - 9 (each thread of middle scan block)


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

// SIMT Kogge-Stone scan kernel
__device__ __inline__ void scan_warp(volatile float* input, int indx = threadIdx.x){
    int lane = indx & 31;
    
    if (lane >= 1)  input[indx] = input[indx - 1] + input[indx];
    if (lane >= 2)  input[indx] = input[indx - 2] + input[indx];
    if (lane >= 4)  input[indx] = input[indx - 4] + input[indx];
    if (lane >= 8)  input[indx] = input[indx - 8] + input[indx];
    if (lane >= 16) input[indx] = input[indx - 16] + input[indx];
}

// SIMT Brent-Kung scan kernel - same as the merrill_srts reduction kernel but since it's the same as the warp size -> no need for __syncthreads()
// BUT BUT!!!!! since this is SIMT -> there is actually 0 gain from reducing the number of operations , so the scan-warp will be used.
__device__ __inline__ void reduce_warp(float* input, int indx = threadIdx.x){
    scan_warp(input, indx);
}

// merrill_srts reduce kernel
__global__ void reduce(float4 *d_input, float *d_output){

    __shared__ float s_data[BLOCKSIZE * 2];//1 cell per thread + another blockdim for easier indx management - will have 2 way bank conflicts though
    
    int idx = blockDim.x * blockIdx.x * REDUCTION_STEPS + threadIdx.x;
    
    d_input += idx;
    d_output += blockIdx.x;

    float4 item;
    float sum = 0;

    #pragma unroll
    for(int i = 0; i < REDUCTION_STEPS; i++){
        item = d_input[i * BLOCKSIZE];
        sum += item.w + item.x + item.y + item.z;
    }

    s_data[threadIdx.x] = sum;

    __syncthreads();

    // we reduce and put the result on the second half of shared memory
    // why no serial reduce like the scan? 
    float *a = s_data;

    #pragma unroll
    for(int d = LOG2_BLOCKSIZE; d > 5; d--){ // 9 -> 5
        
        if( threadIdx.x < (1 << (d - 1)) ){
            a[(1 << d) + threadIdx.x] = a[2 * threadIdx.x] + a[2 * threadIdx.x + 1];
        }

        a = &a[(1 << d)];
        __syncthreads();

    }

    if((threadIdx.x >> 5) == 0){ // warp 0
        reduce_warp(a); // sum will be at idx 31
    }

    // output the sum
    if(threadIdx.x == 31){
        d_output[0] = a[31];
    }

}

// the only change is how smem is handled after the serial scan
__device__ __inline__ void scan_warp_merrill_srts(volatile float (*s_data)[SCAN_SMEM_WIDTH + 1 + 1], int indx = threadIdx.x){
    int lane = indx & 31;

    if (lane >= 1)  s_data[indx][SCAN_SMEM_WIDTH] = s_data[indx - 1][SCAN_SMEM_WIDTH] + s_data[indx][SCAN_SMEM_WIDTH];
    if (lane >= 2)  s_data[indx][SCAN_SMEM_WIDTH] = s_data[indx - 2][SCAN_SMEM_WIDTH] + s_data[indx][SCAN_SMEM_WIDTH];
    if (lane >= 4)  s_data[indx][SCAN_SMEM_WIDTH] = s_data[indx - 4][SCAN_SMEM_WIDTH] + s_data[indx][SCAN_SMEM_WIDTH];
    if (lane >= 8)  s_data[indx][SCAN_SMEM_WIDTH] = s_data[indx - 8][SCAN_SMEM_WIDTH] + s_data[indx][SCAN_SMEM_WIDTH];
    if (lane >= 16) s_data[indx][SCAN_SMEM_WIDTH] = s_data[indx - 16][SCAN_SMEM_WIDTH] + s_data[indx][SCAN_SMEM_WIDTH];

}

// merrill_srts scan kernel
__global__ void scan(float4 *d_input, float *seeds, float4 *d_output){

    __shared__ float s_data[32 * SCAN_STEPS][SCAN_SMEM_WIDTH + 1 + 1]; // 1 for no bank conflict and another one for the result of the warp scan

    int idx = blockDim.x * blockIdx.x * SCAN_STEPS + threadIdx.x;

    d_input += idx;
    d_output += idx;

    int row = threadIdx.x >> LOG2_SCAN_SMEM_WIDTH;
    int col = threadIdx.x & (SCAN_SMEM_WIDTH - 1);

    float4 item[SCAN_STEPS];

    #pragma unroll
    for(int i = 0; i < SCAN_STEPS; i++){
        item[i] = d_input[i * BLOCKSIZE];
        if(threadIdx.x == 0 && i==0 && blockIdx.x > 0){
            item[i].x += seeds[blockIdx.x - 1];
        }
        item[i].y += item[i].x;
        item[i].z += item[i].y;
        item[i].w += item[i].z;
        s_data[32 * i + row][col] = item[i].w;
    }

    __syncthreads();

    // serial reduce
    // each warp going to do 32 rows of smem
    if((threadIdx.x >> 5) < SCAN_STEPS){
        #pragma unroll
        for(int i = 1; i < SCAN_SMEM_WIDTH; i++){
            s_data[threadIdx.x][i] += s_data[threadIdx.x][i - 1];
        }

        scan_warp_merrill_srts(s_data);
    }

    __syncthreads();

    // add the SIMT scan seeds

    #pragma unroll
    for(int i = 0; i < SCAN_STEPS; i++){
        // sum last column of simt scan
        if(threadIdx.x >= SCAN_SMEM_WIDTH){
            item[i].x += s_data[32 * i + row - 1][SCAN_SMEM_WIDTH];
            item[i].y += s_data[32 * i + row - 1][SCAN_SMEM_WIDTH];
            item[i].z += s_data[32 * i + row - 1][SCAN_SMEM_WIDTH];
            item[i].w += s_data[32 * i + row - 1][SCAN_SMEM_WIDTH];
        }
        // sum element before in row, serial scan
        if(threadIdx.x > 0){
            item[i].x += s_data[32 * i + row][col - 1];
            item[i].y += s_data[32 * i + row][col - 1];
            item[i].z += s_data[32 * i + row][col - 1];
            item[i].w += s_data[32 * i + row][col - 1];
        }
        // sum last element of previous simt scan
        if(i > 0){
            item[i].x += s_data[32 * (i - 1) + 31][SCAN_SMEM_WIDTH];
            item[i].y += s_data[32 * (i - 1) + 31][SCAN_SMEM_WIDTH];
            item[i].z += s_data[32 * (i - 1) + 31][SCAN_SMEM_WIDTH];
            item[i].w += s_data[32 * (i - 1) + 31][SCAN_SMEM_WIDTH];
        }
        d_output[i * BLOCKSIZE] = item[i];
    }

}

// two level reduce then scan - middle scan kernel
__global__ void middle_scan(float *seeds){

    __shared__ float s_data[32][SCAN_SMEM_WIDTH + 1 + 1]; // 1 for no bank conflict and another one for the result of the warp scan

    int row = threadIdx.x >> LOG2_SCAN_SMEM_WIDTH;
    int col = threadIdx.x & (SCAN_SMEM_WIDTH - 1);
    
    float seed = 0;
    seeds += threadIdx.x;
    
    // cyclically scan the reduced sums
    #pragma unroll
    for(int i = 0; i < MIDDLE_SCAN_STEP; i++){
        s_data[row][col] = seeds[i * BLOCKSIZE];

        if(threadIdx.x == 0){
            s_data[0][0] += seed;
        }

        __syncthreads();


        if((threadIdx.x >> 5) == 0){
            #pragma unroll
            for(int j = 1; j < SCAN_SMEM_WIDTH; j++){
                s_data[threadIdx.x][j] += s_data[threadIdx.x][j - 1];
            }

            __syncthreads();

            scan_warp_merrill_srts(s_data);
        }

        if(threadIdx.x == 0){
            seed = s_data[31][SCAN_SMEM_WIDTH];
        }

        __syncthreads();

        if(threadIdx.x >= SCAN_SMEM_WIDTH){
            seeds[i * BLOCKSIZE] = s_data[row][col] + s_data[row - 1][SCAN_SMEM_WIDTH];
        } else {
            seeds[i * BLOCKSIZE] = s_data[0][threadIdx.x];
        }
    }
}

// main + interface

void cuda_interface_scan(float4* d_input, float4* d_output){

    int temp = (ARR_SIZE >> 3)/BLOCKSIZE; // each thread processes 2 float4
    dim3 dimBlock(BLOCKSIZE);
    dim3 dimGrid(temp);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float total_time = 0;
    float elapsed_time;

    float *d_scan;
    hipMalloc((void **)&d_scan, temp * sizeof(float));

    hipEventRecord(start, 0);
    reduce<<<dimGrid, dimBlock>>>(d_input, d_scan);
    checkCudaErrors(hipGetLastError());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf( "reduce: %.8f ms\n", elapsed_time);
    total_time += elapsed_time;

    /*std::cout<<"------------------\n";
    float *h_scan = (float*)malloc(temp * sizeof(float));
    hipMemcpy(h_scan, d_scan,  temp * sizeof(float), hipMemcpyDeviceToHost);
    for(int i=510; i < 515; i++)
        std::cout<<h_scan[i]<<"\n";*/

    hipEventRecord(start, 0);
    middle_scan<<<1, dimBlock>>>(d_scan);
    checkCudaErrors(hipGetLastError());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf( "middle scan: %.8f ms\n", elapsed_time);
    total_time += elapsed_time;

    /*std::cout<<"--------------------------middle scan\n";
    hipMemcpy(h_scan, d_scan,  temp * sizeof(float), hipMemcpyDeviceToHost);
    for(int i=510; i < 515; i++)
        std::cout<<h_scan[i]<<"\n";*/

    hipEventRecord(start, 0);
    scan<<<dimGrid, dimBlock>>>(d_input, d_scan, d_output);
    checkCudaErrors(hipGetLastError());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf( "final scan: %.8f ms\n", elapsed_time);
    total_time += elapsed_time;

    printf("total time GPU %.8fms\n", total_time);

    hipFree(d_scan);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}   


void fill_array(float4 *h_input){

    float *temp = (float*) h_input;
    for(int i = 0; i <  ARR_SIZE; i++){
        temp[i] = (float) rand() / RAND_MAX;
    }
}

int main(void){

    srand(0);

    float4 *h_input, *h_output;
    float4 *d_input, *d_output;

    h_input = (float4*) malloc(ARR_SIZE * sizeof(float));
    h_output = (float4*) malloc(ARR_SIZE * sizeof(float));
     
    fill_array(h_input);

    for(int i = 0; i < 5; i++){
        std::cout<<h_input[i].x<<" "<<h_input[i].y<<" "<<h_input[i].z<<" "<<h_input[i].w<<"\n";
    }

    std::cout<<"----------------------\n";

    hipMalloc((void **)&d_input, ARR_SIZE * sizeof(float));
    hipMalloc((void **)&d_output, ARR_SIZE * sizeof(float));

    hipMemcpy(d_input, h_input, ARR_SIZE * sizeof(float), hipMemcpyHostToDevice);

    cuda_interface_scan(d_input, d_output);

    hipMemcpy(h_output, d_output,  ARR_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    std::cout<<"--------GPU----------\n";

    for(int i = 0; i < 5; i++){
        std::cout<<h_output[i].x<<" "<<h_output[i].y<<" "<<h_output[i].z<<" "<<h_output[i].w<<"\n";
    }

    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);

    return 0;
}