#include "hip/hip_runtime.h"
/*
Reference:
    Parallel Scan for Stream Architectures1
    Duane Merrill Andrew Grimshaw

inclusive scan

*/


#include "scan_header.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

__global__ void reduce(float4 *d_input, float *d_output){

    __shared__ float s_data[BLOCKDIM * 2];//1 cell per thread + another blockdim for easier indx management

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    /*if(threadIdx.x == 1 && blockIdx.x == 0){
        printf("%f %f %f %f\n", d_input->w, d_input->x, d_input->y, d_input->z);
    }*/

    d_input += idx;
    d_output += blockIdx.x;

    /*if(threadIdx.x == 1 && blockIdx.x == 0){
        printf("%f %f %f %f\n", d_input->w, d_input->x, d_input->y, d_input->z);
    }*/
    float4 item = *d_input;
    float sum = item.w + item.x + item.y + item.z;

    s_data[threadIdx.x] = sum;

    __syncthreads();

    // we reduce and put the result on the second half of shared memory

    float *a = s_data;

    #pragma unroll
    for(int d = LOG2_BLOCKDIM; d > 0; d--){

        if( threadIdx.x < (1 << (d - 1)) ){
            a[(1 << d) + threadIdx.x] = a[2 * threadIdx.x] + a[2 * threadIdx.x + 1];
        }

        a = &a[(1 << d)];
        __syncthreads();

    }

    // output the sum
    if(threadIdx.x == 0){
        d_output[0] = a[0];
    }
}

// 1 block
__global__ void middle_scan(float *d_input, int iter_per_thread){

    __shared__ float s_data[BLOCKDIM * 2];
    
    float seed = 0;

    if(threadIdx.x == 0){
        seed = d_input[0];
    }
     
    d_input += threadIdx.x;

    // cyclically scan, with the result of each scan becoming the seed to the next
    #pragma unroll
    for(int batch = 0; batch < iter_per_thread; batch++){

        s_data[threadIdx.x] = d_input[batch * iter_per_thread];

        __syncthreads();


        //upsweep
        float *a = s_data;

        #pragma unroll
        for(int d = LOG2_BLOCKDIM; d > 1; d--){ // we don't need last sum, inclusive scan so, the seed = first element

            if( threadIdx.x < (1 << (d - 1)) ){

                a[(1 << d) + threadIdx.x] = a[2 * threadIdx.x] + a[2 * threadIdx.x + 1];

            }

            a += (1 << d);
            __syncthreads();

        }

        if(threadIdx.x == 0){
            a[1] = a[0];
            a[0] = seed; 
        }
        __syncthreads();


        // downsweep
        #pragma unroll
        for(int d = 2; d <= LOG2_BLOCKDIM; d++){

            a -= (1 << d);
            
            if( threadIdx.x < (1 << (d - 1)) ){

                a[2 * threadIdx.x + 1] = a[2 * threadIdx.x] + a[(1 << d) + threadIdx.x];
                a[2 * threadIdx.x] = a[(1 << d) + threadIdx.x];

            }

        __syncthreads();
        }


        d_input[batch * iter_per_thread] = s_data[threadIdx.x];

        if(threadIdx.x == 0){
            seed = s_data[BLOCKDIM - 1]; 
            //printf("%f\n", seed);
        }
    }
}


void scan(float4* d_input, float4* d_output, int arr_size){

    int temp = ((arr_size >> 2) + BLOCKDIM - 1)/BLOCKDIM; // each thread processes 1 float4
    dim3 dimBlock(BLOCKDIM);
    dim3 dimGrid(temp);

    float *d_scan;
    hipMalloc((void **)&d_scan, temp * sizeof(float));

    reduce<<<dimGrid, dimBlock>>>(d_input, d_scan);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    middle_scan<<<1, dimBlock>>>(d_scan, temp/BLOCKDIM);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    float *h_output = (float*) malloc(temp * sizeof(float));
    hipMemcpy(h_output, d_scan, temp * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << temp << "\n";
    std::cout << temp/BLOCKDIM << "\n";

    for(int i = 0; i < 5; i++){
        std::cout << h_output[i] << " ";
    }

    hipFree(d_scan);
}   


void fill_array(float4 *h_input, int arr_size){

    //float *temp = (float*) h_input;
    for(int i = 0; i < arr_size/4; i++){
        //temp[i] = rand() & 15;
        //temp[i] = 1;
        h_input[i].w = 1;
        h_input[i].x = 2;
        h_input[i].y = 3;
        h_input[i].z = 4;
    }
}


int check_solution(float4 *h_input, float4 *h_output, int arr_size){
    float *temp, *h_input1, *h_output1;

    h_input1 = (float*)h_input;
    h_output1 = (float*)h_output;

    temp = (float*) malloc(arr_size * sizeof(float));
    
    temp[0] = 0;
    for(int i = 1; i < arr_size; i++){
        temp[i] = temp[i - 1] + h_input1[i - 1];
    }

    int correct = 1;
    for(int i = 0; i < arr_size; i++){
        if(temp[i] != h_output1[i]){
            correct = 0;
            break;
        }
    }

    return correct;
}


int main(void){

    srand(0);


    float4 *h_input;
    float4 *d_input;
    int arr_size = 1 << 25;

    h_input = (float4*) malloc(arr_size * sizeof(float));

    fill_array(h_input, arr_size);

    hipMalloc((void **)&d_input, arr_size * sizeof(float));
    hipMemcpy(d_input, h_input, arr_size * sizeof(float), hipMemcpyHostToDevice);

    scan(d_input, NULL, arr_size);

    hipDeviceSynchronize();

    hipFree(d_input);
    free(h_input);

    return 0;
}