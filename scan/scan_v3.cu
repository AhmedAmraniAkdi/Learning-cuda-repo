#include "hip/hip_runtime.h"
// we continue the optimisation of scan_v2, now using warp level primitives as shuffle maybe some cooperative groups here and there who knows


#define ARR_SIZE (1 << 25)
#define BLOCKSIZE 128
#define LOG2_BLOCKSIZE 7
#define WARPS_NUM (BLOCKSIZE/32)
#define WORK_PER_THREAD 8 // 8 float4 each thread on scanning and reducing
#define LOG2_WORK_PER_THREAD 5 // 8 elements (2 float4)
#define MIDDLE_SCAN_STEP_PER_THREAD 64 // 2^(25 - 5 - 7 - 7) // -5 (8 float4 loads) - 7 (blocksize) - 7 (each thread of middle scan block)
#define MIDDLE_SCAN_STEP_PER_WARP 8 // 64 steps for 128 threads = 64 steps for 4 warps, 64/(32/4) = 8 iterations of smem warp scan and in each one doing 8 elements
#define MIDDLE_SCAN_WORK_PER_THREAD (BLOCKSIZE/WARPS_NUM) // as we said 8


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

__device__ __inline__ void warp_smem_scan(volatile float *s_data, int indx = threadIdx.x){
    int lane = indx & 31;
    if (lane >= 1)  s_data[indx] = s_data[indx - 1] + s_data[indx];
    if (lane >= 2)  s_data[indx] = s_data[indx - 2] + s_data[indx];
    if (lane >= 4)  s_data[indx] = s_data[indx - 4] + s_data[indx];
    if (lane >= 8)  s_data[indx] = s_data[indx - 8] + s_data[indx];
    if (lane >= 16) s_data[indx] = s_data[indx - 16] + s_data[indx];
}


__global__ void reduce(float4 *d_input, float *d_output){

    __shared__ float s_data[32];
    int idx = blockDim.x * blockIdx.x * WORK_PER_THREAD + threadIdx.x;
    int lane = threadIdx.x & 31;
    int warpid = threadIdx.x >> 5;

    d_input += idx;
    d_output += blockIdx.x;

    float4 item[WORK_PER_THREAD];
    float sum[WORK_PER_THREAD];

    #pragma unroll
    for(int i = 0; i < WORK_PER_THREAD; i++){
        item[i] = d_input[i * BLOCKSIZE];
    }

    #pragma unroll
    for(int i = 0; i < WORK_PER_THREAD; i++){
        sum[i] = item[i].x + item[i].y + item[i].z + item[i].w; 
    }

    #pragma unroll
    for(int i = 0; i < WORK_PER_THREAD; i++){
        sum[i] += __shfl_sync(0xffffffff, sum[i], threadIdx.x - 1);
        sum[i] += __shfl_sync(0xffffffff, sum[i], threadIdx.x - 2);
        sum[i] += __shfl_sync(0xffffffff, sum[i], threadIdx.x - 4);
        sum[i] += __shfl_sync(0xffffffff, sum[i], threadIdx.x - 8);
        sum[i] += __shfl_sync(0xffffffff, sum[i], threadIdx.x - 16);
    }

    //the 31th threads will have the sum - no need for smem

    if(lane == 31){
        #pragma unroll
        for(int i = 0; i < WORK_PER_THREAD; i++){
            s_data[warpid + i * WARPS_NUM] = sum[i];
        }
    }

    __syncthreads();

    if(warpid == 0){
        warp_smem_scan(s_data);
    }

    if(threadIdx.x == 0){
        d_output[0] = s_data[31];
    }
}

// merrill_srts scan kernel
__global__ void scan(float4 *d_input, float *seeds, float4 *d_output){

    __shared__ float s_data[32];
    int idx = blockDim.x * blockIdx.x * WORK_PER_THREAD + threadIdx.x;
    int lane = threadIdx.x & 31;
    int warpid = threadIdx.x >> 5;

    d_input += idx;
    d_output += idx;

    float4 item[WORK_PER_THREAD];

    item[0] = d_input[0];
    item[0].x += (threadIdx.x == 0 && blockIdx.x > 0) ? seeds[blockIdx.x - 1] : 0;
    item[0].y += item[0].x;
    item[0].z += item[0].y;
    item[0].w += item[0].z;
    
    #pragma unroll
    for(int i = 1; i < WORK_PER_THREAD; i++){
        item[i] = d_input[i * BLOCKSIZE];
        item[i].y += item[i].x;
        item[i].z += item[i].y;
        item[i].w += item[i].z;
    }
    
    #pragma unroll
    for(int i = 0; i < WORK_PER_THREAD; i++){
        if (lane >= 1)  item[i] += __shfl_sync(0xffffffff, item[i].w, threadIdx.x - 1);
        if (lane >= 2)  item[i] += __shfl_sync(0xffffffff, item[i].w, threadIdx.x - 2);
        if (lane >= 4)  item[i] += __shfl_sync(0xffffffff, item[i].w, threadIdx.x - 4);
        if (lane >= 8)  item[i] += __shfl_sync(0xffffffff, item[i].w, threadIdx.x - 8);
        if (lane >= 16) item[i] += __shfl_sync(0xffffffff, item[i].w, threadIdx.x - 16);
    }

    if(lane == 31){
        #pragma unroll
        for(int i = 0; i < WORK_PER_THREAD; i++){
            s_data[warpid + i * WARPS_NUM] = item[i].w;
        }
    }

    __syncthreads();

    if(warpid == 0){
        warp_smem_scan(s_data);
    }

    __syncthreads();

    if(warpid > 0){
        #pragma unroll
        for(int i = 0; i < WORK_PER_THREAD; i++){
            item[i] += s_data[warpid - 1 + i * WARPS_NUM];
        }
    }

    #pragma unroll
    for(int i = 0; i < WORK_PER_THREAD; i++){
        d_output[i * BLOCKSIZE] = item[i];
    }
    
}

// two level reduce then scan - middle scan kernel
__global__ void middle_scan(float *seeds){

    __shared__ float s_data[32];
    int lane = threadIdx.x & 31;
    int warpid = threadIdx.x >> 5;
    
    float seed = 0;
    seeds += threadIdx.x;

    float item[MIDDLE_SCAN_WORK_PER_THREAD]; // 4 warps, need 8 to fill 32

    // cyclically scan the reduced sums
    #pragma unroll
    for(int k = 0; k < MIDDLE_SCAN_STEP_PER_WARP; k++){

        item[0] = seeds[0] + seed;
        
        #pragma unroll
        for(int i = 1; i < MIDDLE_SCAN_WORK_PER_THREAD; i++){
            item[i] = seeds[i * BLOCKSIZE];
        }

        #pragma unroll
        for(int i = 0; i < MIDDLE_SCAN_WORK_PER_THREAD; i++){
            if (lane >= 1)  item[i] += __shfl_sync(0xffffffff, item[i], threadIdx.x - 1);
            if (lane >= 2)  item[i] += __shfl_sync(0xffffffff, item[i], threadIdx.x - 2);
            if (lane >= 4)  item[i] += __shfl_sync(0xffffffff, item[i], threadIdx.x - 4);
            if (lane >= 8)  item[i] += __shfl_sync(0xffffffff, item[i], threadIdx.x - 8);
            if (lane >= 16) item[i] += __shfl_sync(0xffffffff, item[i], threadIdx.x - 16);
        }

        if(lane == 31){
            #pragma unroll
            for(int i = 0; i < MIDDLE_SCAN_WORK_PER_THREAD; i++){
                s_data[warpid + i * WARPS_NUM] = item[i];
            }
        }

        __syncthreads();

        if(warpid == 0){
            warp_smem_scan(s_data);
        }

        if(threadIdx.x == 0){
            seed = s_data[31];
        }

        __syncthreads();

        if(warpid > 0){
            #pragma unroll
            for(int i = 0; i < MIDDLE_SCAN_WORK_PER_THREAD; i++){
                item[i] += s_data[warpid - 1 + i * WARPS_NUM];
            }
        }

        #pragma unroll
        for(int i = 0; i < MIDDLE_SCAN_WORK_PER_THREAD; i++){
            seeds[i * BLOCKSIZE] = item[i];
        }

        seeds += BLOCKSIZE/WARPS_NUM * BLOCKSIZE;
    }
}

// main + interface
void cuda_interface_scan(float4* d_input, float4* d_output){

    int temp = ARR_SIZE >> (LOG2_WORK_PER_THREAD + LOG2_BLOCKSIZE); // each thread processes 8 float4
    dim3 dimBlock(BLOCKSIZE);
    dim3 dimGrid(temp);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float total_time = 0;
    float elapsed_time;

    float *d_scan;
    hipMalloc((void **)&d_scan, temp * sizeof(float));

    hipEventRecord(start, 0);
    reduce<<<dimGrid, dimBlock>>>(d_input, d_scan);
    checkCudaErrors(hipGetLastError());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf( "reduce: %.8f ms\n", elapsed_time);
    total_time += elapsed_time;
    
    hipEventRecord(start, 0);
    middle_scan<<<1, dimBlock>>>(d_scan);
    checkCudaErrors(hipGetLastError());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf( "middle scan: %.8f ms\n", elapsed_time);
    total_time += elapsed_time;

    hipEventRecord(start, 0);
    scan<<<dimGrid, dimBlock>>>(d_input, d_scan, d_output);
    checkCudaErrors(hipGetLastError());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf( "final scan: %.8f ms\n", elapsed_time);
    total_time += elapsed_time;

    printf("total time GPU %.8fms\n", total_time);

    hipFree(d_scan);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}   

void fill_array(float4 *h_input){

    float *temp = (float*) h_input;
    for(int i = 0; i <  ARR_SIZE; i++){
        temp[i] = (float) rand() / RAND_MAX;
    }
}

void check(float4 *h_input, float4 *h_output){
    float *temp1 = (float*) h_input;
    float *temp2 = (float*) h_output;
    float *temp3 = (float*) malloc(ARR_SIZE * sizeof(float));

    temp3[0] = temp1[0];
    for(int i = 1; i < ARR_SIZE; i++){
        temp3[i] = temp1[i] + temp3[i - 1];
    }

    std::cout<<"first 1050 elements:\n";
    std::cout<<"element"<<"\tcpu"<<"\tgpu\n";

    for(int i = 0; i < 1050; i++){
        std::cout<<i<<"\t"<<temp1[i] << "\t" << temp3[i] << "\t" << temp2[i] <<"\n";
    }

    free(temp3);
}

int main(void){

    srand(0);

    float4 *h_input, *h_output;
    float4 *d_input, *d_output;

    h_input = (float4*) malloc(ARR_SIZE * sizeof(float));
    h_output = (float4*) malloc(ARR_SIZE * sizeof(float));
     
    fill_array(h_input);

    hipMalloc((void **)&d_input, ARR_SIZE * sizeof(float));
    hipMalloc((void **)&d_output, ARR_SIZE * sizeof(float));

    hipMemcpy(d_input, h_input, ARR_SIZE * sizeof(float), hipMemcpyHostToDevice);

    cuda_interface_scan(d_input, d_output);

    hipMemcpy(h_output, d_output,  ARR_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    check(h_input, h_output);

    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);

    return 0;
}