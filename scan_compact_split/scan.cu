#include "scan_header.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>


__device__ void sweeps(float *d_input, float *intermediate_sum, float *d_output, int step){

}

__device__ void scanBlockSums(float *d_input, float *d_output, int step){

}

__device__ void sumScannedSum(float *scanned_sum, float *output, int step){

}

void scan(float* d_input, float* d_output, int arr_size){

    int temp = (arr_size/sizeof(float) + STEP * BLOCKDIM - 1)/STEP/BLOCKDIM;
    dim3 dimBlock(BLOCKDIM);
    dim3 dimGrid(temp);

    float *intermediate_sum;
    float *intermediate_sum_scanned;
    hipMalloc((void **)&intermediate_sum, arr_size);
    hipMalloc((void **)&intermediate_sum_scanned, arr_size);

    sweeps<<<dimGrid, dimBlock>>>(d_input, intermediate_sum, d_output, STEP);
    hipDeviceSynchronize();

    int new_step = (temp + BLOCKDIM - 1)/BLOCKDIM;

    scanBlockSums<<<1, BLOCKDIM>>>(intermediate_sum, intermediate_sum_scanned, new_step);
    hipDeviceSynchronize();

    sumScannedSum<<<dimGrid, dimBlock>>>(intermediate_sum_scanned, d_output, STEP);
    hipDeviceSynchronize();

    hipFree(intermediate_sum);
    hipFree(intermediate_sum_scanned);
}   

void fill_array(float *h_input, int arr_size){
    for(int i = 0; i < arr_size; i++){
        h_input[i] = rand() % 10;
    }
}

int check_solution(float *h_input, float * h_output, int arr_size){
    float *temp;
    temp = (float*) malloc(arr_size);
    
    temp[0] = 0;
    for(int i = 1; i < arr_size/sizeof(float); i++){
        temp[i] = temp[i - 1] + h_input[i - 1];
    }

    int correct = 1;
    for(int i = 0; i < arr_size/sizeof(float); i++){
        if(temp[i] != h_output[i]){
            correct = 0;
            break;
        }
    }

    return correct;
}


int main(void){

    srand(0);

    float *h_input, *h_output;
    float *d_input, *d_output;
    int arr_size = SIZE_ARRAY * sizeof(float);

    h_input = (float*) malloc(arr_size);
    d_output = (float*) malloc(arr_size);

    fill_array(h_input, arr_size);

    hipMalloc((void **)&d_input, arr_size);
    hipMalloc((void **)&d_output, arr_size);

    hipMemcpy(d_input, h_input, arr_size, hipMemcpyHostToDevice);

    scan(d_input, d_output, arr_size);

    hipDeviceSynchronize();

    hipMemcpy(h_output, d_output, arr_size, hipMemcpyDeviceToHost);

    int correct = check_solution(h_input, h_output, arr_size);

    if(correct){
        std::cout<<"\nCorrect";
    } else {
        std::cout<<"\nNot Correct";
    }

    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);

    return 0;
}