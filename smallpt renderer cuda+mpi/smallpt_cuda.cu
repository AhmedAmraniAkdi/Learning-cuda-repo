#include "hip/hip_runtime.h"
// https://github.com/vchizhov/smallpt-explained/blob/master/smallpt_explained.cpp

// problem: stack overflow on gpu, too many recursion calls
// solution: make radiance function iterative: have a queue where u expand rays (bcs we have reflection+refraction), have it with hardcoded limit
// what is implemented: iterative version in case of refl+refrac, just have a probability it will take 1 or the other!

// the more elegant code is to have different kernels, also more work per thread, etc. and so on, maybe later

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#include <stdio.h>

#include "spheres_rays.cuh"
#include "radiance.cuh"


#define W 1024
#define H 768
#define samps 1024 // samples per subpixel

#define BLOCKDIMX 32
#define BLOCKDIMY 2
#define XSTEP 1

//https://forums.developer.nvidia.com/t/hiprand-init-sequence-number-problem/56573 however xorwow is half the time of philox
__global__ void smallpt_kernel(float3 *d_img, /*hiprandStatePhilox4_32_10_t*/ hiprandState_t *state, float3 cx, float3 cy, Ray cam){

    #pragma unroll
    for(int step = 0; step < XSTEP; step++){

        int idx = blockIdx.x * blockDim.x * XSTEP + threadIdx.x + step * BLOCKDIMX;
        int idy = blockIdx.y * blockDim.y + threadIdx.y;

        int id = idy * W + idx;

        if(idx >= W || idy >= H) return;

        int i = (H - idy - 1 ) * W + idx; // img comes reversed

        if(step == 0) {
            hiprand_init(id, 0, 0, &state[id]);
        }

        float3 r = make_float3(0);

        float3 acum = make_float3(0);

        #pragma unroll
        for(int sy = 0; sy < 2; sy++){

            #pragma unroll
            for(int sx = 0; sx < 2; sx++, r = make_float3(0)){

                #pragma unroll
                for(int s = 0; s < samps ; s++){// each sample is independent, can have another grid doing samps/2 and then atomic sum

                    float r1 = 2 * hiprand_uniform (&state[id]);
                    float dx = r1 < 1 ? sqrt(r1) - 1 : 1 - sqrt(2 - r1);
                    float r2 = 2 * hiprand_uniform (&state[id]);
                    float dy = r2 < 1 ? sqrt(r2) - 1 : 1 - sqrt(2 - r2);

                    float3 d = cx * (((sx + .5 + dx) / 2 + idx) / W - .5) +
                        cy * (((sy + .5 + dy) / 2 + idy) / H - .5) + cam.dir;

                    r = r + radiance(Ray(cam.origin + d * 140, normalize(d)), state, id) * (1./samps);

                }
                acum = acum + clamp(r, 0, 1) * 0.25;
            }
        }

        d_img[i] = acum;

    }

}


int main(){

    // https://en.wikipedia.org/wiki/Ray_tracing_(graphics)#Calculate_rays_for_rectangular_viewport

    Ray cam(make_float3(50, 52 , 295.6), normalize(make_float3(0, -0.042612, -1)));

    float aspectRatio = W/H;

    float vfov = 0.502643;
    float fovScale = 2 * tan(0.5*vfov);

    float3 cx = make_float3(aspectRatio, 0, 0) * fovScale;
    float3 cy = normalize(cross(cx, cam.dir)) * fovScale;

    float3 *h_img = (float3 *)malloc(sizeof(float3) * H * W);
    
	// cuda variables

    /*hiprandStatePhilox4_32_10_t*/ hiprandState_t *devStates;
    hipMalloc((void **)&devStates, sizeof(/*hiprandStatePhilox4_32_10_t*/ hiprandState_t) * W * H );
	checkCudaErrors(hipGetLastError());

    float3 *d_img;
    hipMalloc((void **)&d_img, sizeof(float3) * H * W);
	checkCudaErrors(hipGetLastError());

	hipMemcpyToSymbol(HIP_SYMBOL(spheres), &spheres_cpu, sizeof(spheres_cpu));
	checkCudaErrors(hipGetLastError());

    dim3 dimBlock(BLOCKDIMX, BLOCKDIMY);
    dim3 dimGrid((W + BLOCKDIMX * XSTEP - 1)/BLOCKDIMX/XSTEP, (H + BLOCKDIMY - 1)/BLOCKDIMY);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsed_time;

    hipEventRecord(start, 0);
    smallpt_kernel<<<dimGrid, dimBlock>>>(d_img, devStates, cx, cy, cam);
    checkCudaErrors(hipGetLastError());
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf( "Ray Tracing time: %.8f ms\n", elapsed_time);
    checkCudaErrors(hipGetLastError());

	hipMemcpy(h_img, d_img,  H * W * sizeof(float3), hipMemcpyDeviceToHost);
    checkCudaErrors(hipGetLastError());

	FILE *f = fopen("image.ppm", "w");         // Write image to PPM file.
	fprintf(f, "P3\n%d %d\n%d\n", W, H, 255);
	for (int i = 0; i < W*H; i++)
		fprintf(f, "%d %d %d ", toInt(h_img[i].x), toInt(h_img[i].y), toInt(h_img[i].z));

    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(h_img);
    hipFree(d_img);
    hipFree(devStates);

    return 0;
}