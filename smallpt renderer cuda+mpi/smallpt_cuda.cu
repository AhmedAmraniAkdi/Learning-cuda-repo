#include "hip/hip_runtime.h"
// https://github.com/vchizhov/smallpt-explained/blob/master/smallpt_explained.cpp

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <cuda_gl_interop.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

#define w 1024
#define h 768
#define samps 64

#define BLOCKDIM 128

////

struct Ray{
    float3 origin, dir;
    Ray(float3 origin, float3 dir) : origin(origin), dir(dir) {}
};

////





int main(){

    // https://en.wikipedia.org/wiki/Ray_tracing_(graphics)#Calculate_rays_for_rectangular_viewport

    Ray cam(make_float3(50, 52 , 295.6), normalize(make_float3(0, -0.042612, -1)));

    float aspectRatio = w/h;

    float vfov = 0.502643;
    float fovScale = 2 * tan(0.5*vfov);

    float3 cx = make_float3(aspectRatio, 0, 0) * fovScale;
    float3 cy = normalize(cross(cx, cam.dir)) * fovScale;

    float3 r;
    float3 h_img[w*h];
    memset(h_img, 0, sizeof(float3) * h * w);

    // cuda variables

    hiprandState_t *devStates;
    hipMalloc((void **)&devStates, sizeof(hiprandState) * h * w);

    float3 *d_img;
    hipMalloc((void **)&d_img, sizeof(float3) * h * w);

    dim3 dimBlock(BLOCKDIM, BLOCKDIM);
    dim3 dimGrid((w + BLOCKDIM - 1)/BLOCKDIM, (h + BLOCKDIM - 1)/BLOCKDIM);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsed_time;

    hipEventRecord(start, 0);
    smallpt_kernel<<<dimGrid, dimBlock>>>(d_img, devStates);
    checkCudaErrors(hipGetLastError());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf( "Ray Tracing time: %.8f ms\n", elapsed_time);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_img);
    hipFree(devStates);

    return 0;
}