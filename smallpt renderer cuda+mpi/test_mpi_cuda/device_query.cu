#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

void device_query(int my_rank){  
  
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
    printf("hipGetDeviceCount returned %d\n-> %s\n",
            static_cast<int>(error_id), hipGetErrorString(error_id));
    printf("Result = FAIL\n");
    exit(EXIT_FAILURE);
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0) {
        printf("There are no available device(s) that support CUDA\n");
    } else {
        printf("%d - Detected %d CUDA Capable device(s)\n", my_rank, deviceCount);
    }

    int dev, driverVersion = 0, runtimeVersion = 0;

    for (dev = 0; dev < deviceCount; ++dev) {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

        // Console log
        hipDriverGetVersion(&driverVersion);
        hipRuntimeGetVersion(&runtimeVersion);
        printf("%d -  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", my_rank, 
                driverVersion / 1000, (driverVersion % 100) / 10,
                runtimeVersion / 1000, (runtimeVersion % 100) / 10);
        printf("%d -  CUDA Capability Major/Minor version number:    %d.%d\n", my_rank,
                deviceProp.major, deviceProp.minor);
    }
}